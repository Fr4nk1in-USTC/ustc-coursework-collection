#include "hip/hip_runtime.h"
#include <algorithm>
#include <bits/chrono.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <random>

using std::cout;
using std::endl;
using std::equal;
using std::fixed;
using std::mt19937;
using std::random_device;
using std::setprecision;
using std::setw;
using std::uniform_real_distribution;
using std::chrono::duration_cast;

using ns = std::chrono::nanoseconds;
auto now = std::chrono::high_resolution_clock::now;

const float  MIN_FLOAT = 0;
const float  MAX_FLOAT = 1000;
const size_t SCALES[]  = {100000, 200000, 1000000, 2000000, 10000000, 20000000};

const size_t NUM_THREADS = 256;

/**
 * Generate two random vectors `a` and `b` of size `size`.
 */
void generate_vector(float *a, float *b, size_t size)
{
    random_device                    rd;
    mt19937                          gen(rd());
    uniform_real_distribution<float> dis(MIN_FLOAT, MAX_FLOAT);

    for (size_t i = 0; i < size; i++) {
        a[i] = dis(gen);
        b[i] = dis(gen);
    }
}

/**
 * Add two vectors `a` and `b` and store the result in `c`.
 * The caller should ensure that `c` has the memory space.
 */
void vecadd_cpu(const float *a, const float *b, float *c, size_t size)
{
    for (size_t i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ static void vecadd_kernel(const float *a, const float *b, float *c,
                                     size_t size)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

bool verify(const float *a, const float *b, size_t size)
{
    for (size_t i = 0; i < size; i++) {
        if (fabs(a[i] - b[i]) > 1e-5) {
            return false;
        }
    }
    return true;
}

int main()
{
    cout << "┌────────┬──────────────────────┬────────────────┐" << endl
         << "│        │     running time (ms)│         speedup│" << endl
         << "│    size├─────────┬──────┬─────┼─────────┬──────┤" << endl
         << "│        │GPU total│kernel│  CPU│GPU total│kernel│" << endl
         << "├────────┼─────────┼──────┼─────┼─────────┼──────┤" << endl;
    cout << setprecision(2) << fixed;
    for (auto &size : SCALES) {
        float *host_a     = new float[size];
        float *host_b     = new float[size];
        float *host_c_gpu = new float[size];
        float *host_c_cpu = new float[size];

        /* Generate random vector */
        generate_vector(host_a, host_b, size);

        /* Allocate device memory */
        float *device_a, *device_b, *device_c;
        hipMalloc((void **)&device_a, size * sizeof(float));
        hipMalloc((void **)&device_b, size * sizeof(float));
        hipMalloc((void **)&device_c, size * sizeof(float));

        /* Define block and grid size */
        dim3 dim_block(NUM_THREADS, 1, 1);
        int  num_block = (size - 1) / NUM_THREADS + 1;
        dim3 dim_grid(num_block, 1, 1);

        /* Time the GPU additon */
        auto gpu_start = now();
        hipMemcpy(device_a, host_a, size * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(device_b, host_b, size * sizeof(float),
                   hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        auto kernel_start = now();
        vecadd_kernel<<<dim_grid, dim_block>>>(device_a, device_b, device_c,
                                               size);
        hipDeviceSynchronize();
        auto kernel_end = now();

        hipMemcpy(host_c_gpu, device_c, size * sizeof(float),
                   hipMemcpyDeviceToHost);
        auto gpu_end = now();

        /* Time the CPU addition */
        auto cpu_start = now();
        vecadd_cpu(host_a, host_b, host_c_cpu, size);
        auto cpu_end = now();

        /* Verify the result */
        if (!verify(host_c_gpu, host_c_cpu, size)) {
            cout << "Verification failed on size " << size << endl;
            return 1;
        }

        /* Calculate the elapsed time */
        ns gpu_ns    = duration_cast<ns>(gpu_end - gpu_start);
        ns kernel_ns = duration_cast<ns>(kernel_end - kernel_start);
        ns cpu_ns    = duration_cast<ns>(cpu_end - cpu_start);

        float gpu_time    = gpu_ns.count() / 1e6;
        float kernel_time = kernel_ns.count() / 1e6;
        float cpu_time    = cpu_ns.count() / 1e6;

        float speedup        = cpu_time / gpu_time;
        float kernel_speedup = cpu_time / kernel_time;

        cout << "│" << setw(8) << size << "│" << setw(9) << gpu_time << "│"
             << setw(6) << kernel_time << "│" << setw(5) << cpu_time << "│"
             << setw(9) << speedup << "│" << setw(6) << kernel_speedup << "│"
             << endl;

        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);

        delete[] host_a;
        delete[] host_b;
        delete[] host_c_gpu;
        delete[] host_c_cpu;
    }
    cout << "└────────┴─────────┴──────┴─────┴─────────┴──────┘" << endl;
}
