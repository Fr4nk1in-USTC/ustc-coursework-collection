#include <chrono>
#include <cstddef>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <random>

using std::cout;
using std::endl;
using std::equal;
using std::fixed;
using std::mt19937;
using std::random_device;
using std::setprecision;
using std::setw;
using std::uniform_real_distribution;
using std::chrono::duration_cast;

using ns = std::chrono::nanoseconds;
auto now = std::chrono::high_resolution_clock::now;

const float MIN_FLOAT = 0;
const float MAX_FLOAT = 100;

const size_t BLOCK_SIZE     = 32;               // block of 16 * 16 threads
const size_t MATRIX_SIZES[] = {10, 100, 1000};  // matrix size

/**
 * Generate two random vectors `a` and `b` of size `size`.
 */
void generate_matrix(float *a, float *b, size_t size)
{
    random_device                    rd;
    mt19937                          gen(rd());
    uniform_real_distribution<float> dis(MIN_FLOAT, MAX_FLOAT);

    for (size_t i = 0; i < size * size; i++) {
        a[i] = dis(gen);
        b[i] = dis(gen);
    }
}

__global__ static void matmul_kernel(const float *a, const float *b, float *c,
                                     size_t size)
{
    __shared__ float a_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float b_sub[BLOCK_SIZE][BLOCK_SIZE];

    /* Block indices */
    const size_t block_x = blockIdx.x;
    const size_t block_y = blockIdx.y;

    /* Thread indices */
    const size_t thread_x = threadIdx.x;
    const size_t thread_y = threadIdx.y;

    /* Number of blocks */
    const size_t num_sub = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float c_sub = 0;

    for (size_t sub = 0; sub < num_sub; sub++) {
        /* Load the sub-matrices into the shared memory */
        /* Each thread loads one element of each sub-matrix */
        size_t a_x = block_x * BLOCK_SIZE + thread_x;
        size_t a_y = sub * BLOCK_SIZE + thread_y;
        if (a_x < size and a_y < size) {
            a_sub[thread_x][thread_y] = a[a_x * size + a_y];
        } else {
            a_sub[thread_x][thread_y] = 0;
        }

        size_t b_x = sub * BLOCK_SIZE + thread_x;
        size_t b_y = block_y * BLOCK_SIZE + thread_y;
        if (b_x < size and b_y < size) {
            b_sub[thread_x][thread_y] = b[b_x * size + b_y];
        } else {
            b_sub[thread_x][thread_y] = 0;
        }

        __syncthreads();

        /* Multiply the two sub-matrices */
        for (size_t k = 0; k < BLOCK_SIZE; k++) {
            c_sub += a_sub[thread_x][k] * b_sub[k][thread_y];
        }

        __syncthreads();
    }

    size_t c_x = block_x * BLOCK_SIZE + thread_x;
    size_t c_y = block_y * BLOCK_SIZE + thread_y;
    if (c_x < size and c_y < size) {
        c[c_x * size + c_y] = c_sub;
    }
}

void matmul_cpu(const float *a, const float *b, float *c, size_t size)
{
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            float t = 0;
            for (size_t k = 0; k < size; k++) {
                t += a[i * size + k] * b[k * size + j];
            }
            c[i * size + j] = t;
        }
    }
}

bool verify(const float *a, const float *b, size_t size)
{
    for (size_t i = 0; i < size * size; i++) {
        if (fabs(a[i] - b[i]) / a[i] > 1e-6) {
            return false;
        }
    }
    return true;
}

void print_matrix(const float *a, size_t size)
{
    for (size_t i = 0; i < size * size; i++) {
        cout << setw(10) << fixed << setprecision(1) << a[i];
        if ((i + 1) % size == 0) {
            cout << endl;
        }
    }
}

int main()
{
    cout << "┌────┬────────────────────────┬────────────────┐" << endl
         << "│    │       running time (ms)│         speedup│" << endl
         << "│size├─────────┬──────┬───────┼─────────┬──────┤" << endl
         << "│    │GPU total│kernel│    CPU│GPU total│kernel│" << endl
         << "├────┼─────────┼──────┼───────┼─────────┼──────┤" << endl;
    cout << fixed << setprecision(2);
    for (auto &size : MATRIX_SIZES) {
        float *host_a     = new float[size * size];
        float *host_b     = new float[size * size];
        float *host_c_gpu = new float[size * size];
        float *host_c_cpu = new float[size * size];

        /* Generate random matrices. */
        generate_matrix(host_a, host_b, size);

        /* Allocate device memory. */
        float *device_a, *device_b, *device_c;
        hipMalloc((void **)&device_a, size * size * sizeof(float));
        hipMalloc((void **)&device_b, size * size * sizeof(float));
        hipMalloc((void **)&device_c, size * size * sizeof(float));

        /* Define block and grid size */
        int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 dim_grid(grid_size, grid_size, 1);

        /* Time the GPU multiplication */
        auto gpu_start = now();
        hipMemcpy(device_a, host_a, size * size * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(device_b, host_b, size * size * sizeof(float),
                   hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        auto kernel_start = now();
        matmul_kernel<<<dim_grid, dim_block, 0>>>(device_a, device_b, device_c,
                                                  size);
        hipDeviceSynchronize();
        auto kernel_end = now();

        hipMemcpy(host_c_gpu, device_c, size * size * sizeof(float),
                   hipMemcpyDeviceToHost);
        auto cuda_end = now();

        /* Time the CPU multiplication */
        auto cpu_start = now();
        matmul_cpu(host_a, host_b, host_c_cpu, size);
        auto cpu_end = now();

        /* Verify the result */
        if (!verify(host_c_gpu, host_c_cpu, size)) {
            cout << "Verification failed on size " << size << endl;
            cout << "CPU result:" << endl;
            print_matrix(host_c_cpu, size);
            cout << "GPU result:" << endl;
            print_matrix(host_c_gpu, size);
            return 1;
        }

        /* Calculate the elapsed time */
        ns gpu_ns    = duration_cast<ns>(cuda_end - gpu_start);
        ns kernel_ns = duration_cast<ns>(kernel_end - kernel_start);
        ns cpu_ns    = duration_cast<ns>(cpu_end - cpu_start);

        float gpu_time    = gpu_ns.count() / 1e6;
        float kernel_time = kernel_ns.count() / 1e6;
        float cpu_time    = cpu_ns.count() / 1e6;

        float speedup        = cpu_time / gpu_time;
        float kernel_speedup = cpu_time / kernel_time;

        cout << "│" << setw(4) << size << "│" << setw(9) << gpu_time << "│"
             << setw(6) << kernel_time << "│" << setw(7) << cpu_time << "│"
             << setw(9) << speedup << "│" << setw(6) << kernel_speedup << "│"
             << endl;

        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);

        delete[] host_a;
        delete[] host_b;
        delete[] host_c_gpu;
        delete[] host_c_cpu;
    }
    cout << "└────┴─────────┴──────┴───────┴─────────┴──────┘" << endl;
}
